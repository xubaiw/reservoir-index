#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 65536 // 256 ^ 256

#define h2d(h,d,n) hipMemcpy(d,h,sizeof(int)*n, hipMemcpyHostToDevice)
#define d2h(d,h,n) hipMemcpy(h,d,sizeof(int)*n, hipMemcpyDeviceToHost)
#define dma(v,n) int *v; hipMalloc((void **)&v, sizeof(int)*n)
#define dfr(v) hipFree(v)
#define DO(n) for(int i=0;i<n;++i)
#define P(n) for(int i=0;i<n;++i)
#define BIx blockIdx.x
#define BX(v) v[BIx]
#define OP1(nm) __global__ void nm(int *r, int*x)
#define OP2(nm) __global__ void nm(int *r, int *x, int *y)

OP1(not){ BX(r) = ~BX(a); }
OP2(xor){ BX(r) = BX(a) ^ BX(b); }
OP2(and){ BX(r) = BX(a) & BX(b); }

int main() {

  // arrays on the cpu: h:'host'
  int ha[N], hb[N];

  // arrays on the gpu d:'device'
  int *da, *db; dma(da, N); dma(db, N);

  // ha: !N
  DO(N) ha[i] = i;

  dma(II, N);  dma(OO, N);

  h2d(ha, da, N);
  add<<<N, 1>>>(da, db);
  d2h(db, hb, N);

  DO(N) { P("%6x ", hb[i]); if (!(15&i-1)) P("\n"); }
  P("\n");

  dfr(da); dfr(db);

  P("hello from CUDA!\n");
  return 0;
}
