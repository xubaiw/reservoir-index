// based on https://gist.github.com/dpiponi/1502434

#include <hip/hip_runtime.h>
#include <stdio.h>

#define N  256 // 0x1d710 // 65536 // 4096 //1024

#define h2d(h,d,n) hipMemcpy(d,h,sizeof(int)*n, hipMemcpyHostToDevice)
#define d2h(d,h,n) hipMemcpy(h,d,sizeof(int)*n, hipMemcpyDeviceToHost)
#define I(n) for(int i=0;i<n;++i)

__global__
void add(int *a, int *b) { int i = blockIdx.x; if (i<N) { b[i] = 2*a[i]; }}

int main() {

  // arrays on the cpu: h:'host'
  int ha[N], hb[N];

  // arrays on the gpu d:'device'
  int *da, *db;
  hipMalloc((void **)&da, N*sizeof(int));
  hipMalloc((void **)&db, N*sizeof(int));

  // ha: !N
  I(N) ha[i] = i;

  h2d(ha, da, N);
  add<<<N, 1>>>(da, db);
  d2h(db, hb, N);

  for(int i=0; i<N;){ printf("%6x ", hb[i]); if(!(++i&15))printf("\n"); }
  printf("\n");

  hipFree(da);
  hipFree(db);

  printf("hello from CUDA!\n");
  int dc; hipGetDeviceCount(&dc);
  printf("Device count: %d\n", dc);
  return 0;
}
